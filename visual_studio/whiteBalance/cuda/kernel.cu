#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <cstdio>
#include <cmath>
#include <cstring>
#include <vector>
#include <algorithm>
#include <ctime>
#include <thread>
#include <omp.h>

#define SHOW_INFO false
#define OUTPUT_VIDEO true
#define TD_MAX_SIZE 50
#define MAX_ROWS 1920
#define MAX_COLS 1080

using namespace std;
using namespace cv;

int threadNum;
Mat imgs[TD_MAX_SIZE];

VideoWriter setOutput(const VideoCapture &input) {
	// Reference from
	// http://docs.opencv.org/2.4/doc/tutorials/highgui/video-write/video-write.html

	// Acquire input size
	Size S = Size((int)input.get(CV_CAP_PROP_FRAME_WIDTH),
		(int)input.get(CV_CAP_PROP_FRAME_HEIGHT));

	// Get Codec Type- Int form
	int ex = static_cast<int>(input.get(CV_CAP_PROP_FOURCC));

	VideoWriter output;
	output.open("outputVideo.avi", CV_FOURCC('H', 'F', 'Y', 'U'), input.get(CV_CAP_PROP_FPS), S, true);

	return output;
}

__global__ void whiteBa(unsigned char *mat, double avg_r, double avg_g, double avg_b, int tot) {
	
	int id = (blockIdx.x * blockDim.x + threadIdx.x) * 3;
	if (id < tot) {
		double adj_r = avg_g / avg_r;
		double adj_b = avg_g / avg_b;
		mat[id + 2] = mat[id + 2] * adj_r < 255.0 ? mat[id + 2] * adj_r : 255;
		mat[id] = mat[id] * adj_b < 255.0 ? mat[id] * adj_b : 255;
	}
}

void whiteBalance_CUDA(Mat imgs[], const int &sz) {
	
	int i, j, k, rows = imgs[0].rows, cols = imgs[0].cols;
	int totalElements = rows * cols;
	int size = totalElements * sizeof(unsigned char) * 3;
	unsigned char *device_mat = NULL;

	
	for (k = 0; k < sz; k++) {

		double avg_r = 0.0, avg_g = 0.0, avg_b = 0.0;
#pragma omp parallel for private(i, j) reduction(+:avg_r, avg_g, avg_b)
		for (i = 0; i < rows; i++)
			for (j = 0; j < cols; j++) {
				avg_r += imgs[k].at<Vec3b>(i, j)[2];
				avg_g += imgs[k].at<Vec3b>(i, j)[1];
				avg_b += imgs[k].at<Vec3b>(i, j)[0];
			}
		avg_r /= totalElements;
		avg_g /= totalElements;
		avg_b /= totalElements;

		hipMalloc(&device_mat, size);
		hipMemcpy(device_mat, imgs[k].ptr(), size, hipMemcpyHostToDevice);

		int threadsPerBlock = 1024;
		int blocksPerGrid = (totalElements + threadsPerBlock - 1) / threadsPerBlock;
		whiteBa << <blocksPerGrid, threadsPerBlock >> >(device_mat, avg_r, avg_g, avg_b, totalElements);

		hipMemcpy(imgs[k].ptr(), device_mat, size, hipMemcpyDeviceToHost);

		hipFree(device_mat);

	}
}

void inputVideo(const char *filePath, int rank, int sz, int fid) {
	int numPerThread = sz / threadNum;
	int from = rank * numPerThread;
	int to = (rank == threadNum - 1) ? sz : from + numPerThread;
	VideoCapture cpVideo(filePath);
	cpVideo.set(CV_CAP_PROP_POS_FRAMES, fid + from);

	for (int i = from; i<to; ++i)
		cpVideo >> imgs[i];
}

int main(int argc, const char** argv){
	if (CV_MAJOR_VERSION < 3) {
		puts("Advise you update to OpenCV3");
	}
	if (argc<2) {
		puts("Please specify input image path");
		return 0;
	}
	if (argc<3) {
		puts("Please specify thread num");
		return 0;
	}

	VideoCapture captureVideo;
	captureVideo.open(argv[1]);
	if (!captureVideo.isOpened()) {
		puts("Fail to open video");
		return 0;
	}

	// Setup video output
	VideoWriter outputVideo;
	if (OUTPUT_VIDEO)
		outputVideo = setOutput(captureVideo);
	
	threadNum = atoi(argv[2]);
	if (SHOW_INFO)
		printf("threads: %d\n", threadNum);
	outputVideo.set(CV_CAP_PROP_BUFFERSIZE, 1);

	double Calculate = 0, Input = 0, Output = 0;
	double Total = getTickCount(), Last;

	
	int numFrames = captureVideo.get(CV_CAP_PROP_FRAME_COUNT);
	for (int fid = 0; fid<numFrames; fid += TD_MAX_SIZE) {

		int sz = numFrames - fid;
		if (sz > TD_MAX_SIZE) sz = TD_MAX_SIZE;

		// store all thread
		vector<thread> threads;

		// input enough frames
		Last = getTickCount();
		for (int i = 0; i<threadNum; ++i)
			threads.emplace_back(thread(inputVideo, argv[1], i, sz, fid));
		for (int i = 0; i<threadNum; ++i)
			threads[i].join();
		threads.clear();
		Input += getTickCount() - Last;

		// proc all received frames
		Last = getTickCount();
		whiteBalance_CUDA(imgs, sz);
		Calculate += getTickCount() - Last;

		if (OUTPUT_VIDEO) {
			Last = getTickCount();
			for (int i = 0; i<sz; ++i)
				outputVideo << imgs[i];
			Output += getTickCount() - Last;
		}
	}

	Total = getTickCount() - Total;

	printf("    Total: %.3fs (include time count)\n", Total / getTickFrequency());
	printf("    Input: %.3fs\n", Input / getTickFrequency());
	printf("   Output: %.3fs\n", Output / getTickFrequency());
	printf("Calculate: %.3fs\n", Calculate / getTickFrequency());

	return 0;
}
